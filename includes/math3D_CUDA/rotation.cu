#include "hip/hip_runtime.h"
#pragma once

#define INSIDE_ROTATION_CU_FILE 1
#include "rotation.cuh"

#include<math.h>

namespace manipulation3dD {

	__device__ __host__ double toDeg(double rad) { return rad * 180 / pi; }
	__device__ __host__ double toRad(double deg) { return deg * pi / 180; }

	__device__ __host__ vec3d getDir(double yaw, double pitch) {
		vec3d newX = vec3d(cos(yaw), sin(yaw), 0);
		vec3d rval = vec3d::add(vec3d::multiply(newX, cos(pitch)), vec3d::multiply(vec3d::vec3(0, 0, 1), sin(pitch)));
		return rval;
	}

	__device__ __host__ vec3d getRotation(vec3d a,bool *err) {
		vec3d rval;
		if (a.mag2() == 0) {
			*err = true;//error
			return vec3d(0, 0, 0);
		}
		*err = false;//no error
		rval.y = (pi / 2) - vec3d::angleRaw(a, vec3d(0, 0, 1));
		rval.x = vec3d::angleRaw(vec3d(1, 0, 0), vec3d(a.x, a.y, 0));
		if (a.y < 0)rval.x = 2 * pi - rval.x;
		return rval;
	}

	__device__ __host__ vec3d getRotationRaw_s(vec3d a , vec3d defaultRVal) {
		vec3d rval;
		if (a.mag2() == 0) {
			return defaultRVal;
		}
		rval.y = (pi / 2) - vec3d::angleRaw(a, vec3d(0, 0, 1));
		rval.x = vec3d::angleRaw(vec3d(1, 0, 0), vec3d(a.x, a.y, 0));
		if (a.y < 0)rval.x = 2 * pi - rval.x;
		return rval;
	}
	
	__device__ __host__ vec3d getRotationRaw(vec3d a) {
		vec3d rval;
		rval.y = (pi / 2) - vec3d::angleRaw(a, vec3d(0, 0, 1));
		rval.x = vec3d::angleRaw(vec3d(1, 0, 0), vec3d(a.x, a.y, 0));
		if (a.y < 0)rval.x = 2 * pi - rval.x;
		return rval;
	}


	//coordinate system functions

	__device__ __host__ void coordinateSystem::setOrigin(vec3d vec) { origin = vec; }
	__device__ __host__ void coordinateSystem::setAngle(vec3d vec) { angle = vec; reset = true; }
	__device__ __host__ void coordinateSystem::setScale(vec3d vec) { scale = vec; reset = true; }
	__device__ __host__ void coordinateSystem::setAxis(vec3d* Axis) {
		axis[0] = Axis[0];
		axis[1] = Axis[1];
		axis[2] = Axis[2];
		//set scale
		scale = getScale(axis);

		//set angle
		angle = getAngle(axis);
	}

	__device__ __host__ vec3d coordinateSystem::getOrigin() { return origin; }
	__device__ __host__ vec3d coordinateSystem::getAngle() { return angle; }
	__device__ __host__ vec3d coordinateSystem::getScale() { return scale; }
	__device__ __host__ vec3d* coordinateSystem::getAxis() { if (reset)resetAxis(); reset = false; return axis; };

	__device__ __host__ coordinateSystem::coordinateSystem(vec3d Origin, vec3d Rot, vec3d Scale) {
		origin = Origin;
		angle = Rot;
		scale = Scale;
		resetAxis();
	}

	__device__ __host__ void coordinateSystem::resetAxis() {
		axis[0] = getDir(angle.x, angle.y);
		axis[1] = getDir(angle.x + pi / 2, 0);
		axis[2] = getDir(angle.x, angle.y + pi / 2);
		//axis[1] and axis[2] are temp axes
		vec3d cAxis[2];
		cAxis[0] = vec3d::add(vec3d::multiply(axis[1], cos(angle.z)), vec3d::multiply(axis[2], sin(angle.z)));
		cAxis[1] = vec3d::cross(axis[0], cAxis[0]);
		axis[0] = vec3d::multiply(axis[0], scale.x);
		axis[1] = vec3d::multiply(cAxis[0], scale.y);
		axis[2] = vec3d::multiply(cAxis[1], scale.z);
	}

	__device__ __host__ void coordinateSystem::set(coordinateSystem& cs) {
		//set origin
		origin = cs.getOrigin();
		scale = cs.getScale();
		angle = cs.getAngle();
		axis[0] = cs.getAxis()[0];
		axis[1] = cs.getAxis()[1];
		axis[2] = cs.getAxis()[2];
	}

	__device__ __host__ vec3d coordinateSystem::getScale(vec3d* axis) {
		return vec3d(axis[0].mag(), axis[1].mag(), axis[2].mag());
	}

	__device__ __host__ vec3d coordinateSystem::getAngle(vec3d* axis) {
		vec3d rVal;
		rVal = getRotationRaw_s(axis[0]);
		vec3d tempAxis = getDir(angle.x + pi / 2, 0);
		rVal.z = vec3d::angleRaw_s(tempAxis, axis[1]);
		tempAxis = getDir(angle.x, angle.y + pi / 2);
		if (vec3d::dot(tempAxis, axis[1]) < 0)rVal.z *= -1;
		return rVal;
	}

	__device__ __host__ vec3d coordinateSystem::getInCoordinateSystem(vec3d realCoord) {
		if (reset) {
			resetAxis();
			reset = false;
		}
		vec3d rVal;
		realCoord = vec3d::subtract(realCoord, origin);
		rVal.x = vec3d::componentRaw_s(realCoord, axis[0]);
		rVal.y = vec3d::componentRaw_s(realCoord, axis[1]);
		rVal.z = vec3d::componentRaw_s(realCoord, axis[2]);
		vec3d Scale = scale;
		if (Scale.x == 0)Scale.x == 1;
		if (Scale.y == 0)Scale.y == 1;
		if (Scale.z == 0)Scale.z == 1;
		rVal.x /= Scale.x;
		rVal.y /= Scale.y;
		rVal.z /= Scale.z;
		return rVal;
	}

	__device__ __host__ vec3d coordinateSystem::getRealWorldCoordinates(vec3d CSCoord) {
		if (reset) {
			resetAxis();
			reset = false;
		}
		return vec3d::add(origin, vec3d::add(vec3d::multiply(axis[0], CSCoord.x), vec3d::add(vec3d::multiply(axis[1], CSCoord.y), vec3d::multiply(axis[2], CSCoord.z))));
	}

	__device__ __host__ void coordinateSystem::addRelativeRot(vec3d rot) {

		vec3d oldAxis[3];
		vec3d Scale = scale;
		if (Scale.x == 0)Scale.x = 1;
		if (Scale.y == 0)Scale.y = 1;
		if (Scale.z == 0)Scale.z = 1;
		oldAxis[0] = vec3d::multiply(axis[0], 1 / Scale.x);
		oldAxis[1] = vec3d::multiply(axis[1], 1 / Scale.y);
		oldAxis[2] = vec3d::multiply(axis[2], 1 / Scale.z);


		vec3d dir[3];
		dir[0] = getDir(rot.x, rot.y);
		dir[1] = getDir(rot.x + pi / 2, 0/*rot.y*/);
		dir[2] = getDir(rot.x, rot.y + pi / 2);
		{
			vec3d temp[2];
			temp[0] = vec3d::add(vec3d::multiply(dir[1], cos(rot.z)), vec3d::multiply(dir[2], sin(rot.z)));
			temp[1] = vec3d::cross(dir[0], temp[1]);
			dir[1] = temp[0];
			dir[2] = temp[1];
		}

		vec3d newAxis[3];
		newAxis[0] = vec3d::multiply(vec3d::add(vec3d::multiply(oldAxis[0], dir[0].x), vec3d::add(vec3d::multiply(oldAxis[1], dir[0].y), vec3d::multiply(oldAxis[2], dir[0].z))), scale.x);
		newAxis[1] = vec3d::multiply(vec3d::add(vec3d::multiply(oldAxis[0], dir[1].x), vec3d::add(vec3d::multiply(oldAxis[1], dir[1].y), vec3d::multiply(oldAxis[2], dir[1].z))), scale.y);
		newAxis[2] = vec3d::multiply(vec3d::add(vec3d::multiply(oldAxis[0], dir[2].x), vec3d::add(vec3d::multiply(oldAxis[1], dir[2].y), vec3d::multiply(oldAxis[2], dir[2].z))), scale.z);
		setAxis(newAxis);
	}

	__device__ __host__ void coordinateSystem::addRelativePos(vec3d pos) {
		vec3d Axis[3];
		vec3d Scale = scale;
		if (Scale.x == 0)Scale.x = 1;
		if (Scale.y == 0)Scale.y = 1;
		if (Scale.z == 0)Scale.z = 1;
		Axis[0] = vec3d::multiply(axis[0], 1 / Scale.x);
		Axis[1] = vec3d::multiply(axis[1], 1 / Scale.y);
		Axis[2] = vec3d::multiply(axis[2], 1 / Scale.z);

		origin = vec3d::add(origin, vec3d::add(vec3d::multiply(Axis[0], pos.x), vec3d::add(vec3d::multiply(Axis[1], pos.y), vec3d::multiply(Axis[2], pos.z))));
	}

	__device__ __host__ void coordinateSystem::addRotationAboutAxis(vec3d W) {
		transform T;
		vec3d angle = getRotationRaw_s(W);
		T.CS.setAngle(angle);
		vec3d oldAxis[3] = { axis[0],axis[1],axis[2] };
		T.addVec(oldAxis[0], oldAxis);
		T.addVec(oldAxis[1], oldAxis + 1);
		T.addVec(oldAxis[2], oldAxis + 2);
		angle.z = W.mag();
		T.CS.setAngle(angle);
		T.update();
		setAxis(oldAxis);
	}
	
	//transform functions
	__host__ void transform::addVec(vec3d val, vec3d* adress) {
		data.push_back(CS.getInCoordinateSystem(val));
		dataAddress.push_back(adress);
	}

	__host__ void transform::update() {
		for (int i = 0; i < data.size(); ++i) {
			*(dataAddress[i]) = CS.getRealWorldCoordinates(data[i]);
		}
	}

	
}