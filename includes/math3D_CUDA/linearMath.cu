#include "hip/hip_runtime.h"
#pragma once
#define INSIDE_linearMath_CU_FILE 1
#include "linearMath.cuh"

#include <math.h>

namespace linearMathD {//double precesion


	//line class
	__host__ __device__ line::line() {
		setRaw(vec3d(0, 0, 0), vec3d(1, 0, 0));
	}

	__host__ __device__ line::line(vec3d PT, vec3d DR) {
		setRaw_s(PT, DR);
	}

	__host__ __device__ bool line::set(vec3d PT, vec3d DR) {
		pt = PT;
		if (vec3d::isNUL(DR)) {
			dr = vec3d(1, 0, 0);
			return true;//error
		}
		dr = DR;
		return false;
	}

	__host__ __device__ void line::setRaw_s(vec3d PT, vec3d DR) {
		pt = PT;
		if (vec3d::isNUL(DR)) dr = vec3d(1, 0, 0);
		else dr = DR;
	}

	__host__ __device__ void line::setRaw(vec3d PT, vec3d DR) {
		pt = PT;
		dr = DR;
	}

	

	__host__ __device__ void line::setPT(vec3d PT) {
		pt = PT;
	}

	__host__ __device__ bool line::setDR(vec3d DR) {
		if (vec3d::isNUL(DR)) {
			dr = vec3d(1, 0, 0);
			return true;//error
		}
		dr = DR;
		return false;
	}

	__host__ __device__ void line::setDRRaw_s(vec3d DR) {
		if (vec3d::isNUL(DR))dr = vec3d(1, 0, 0);
		else dr = DR;
	}

	__host__ __device__ void line::setDRRaw(vec3d DR) {
		dr = DR;
	}

	__host__ __device__ vec3d line::getPt() { return pt; }
	__host__ __device__ vec3d line::getDr() { return dr; }

	//plane class
	__host__ __device__ plane::plane() {
		setRaw(vec3d(0,0,0),vec3d(1,0,0));
	}

	__host__ __device__ plane::plane(vec3d PT, vec3d DR) {
		setRaw_s(PT, DR);
	}

	__host__ __device__ bool plane::set(vec3d PT, vec3d DR) {
		pt = PT;
		if (vec3d::isNUL(DR)) {
			dr = vec3d(1, 0, 0);
			return true;//error
		}
		dr = DR;
		return false;
	}
	__host__ __device__ void plane::setRaw_s(vec3d PT, vec3d DR) {
		pt = PT;
		if (vec3d::isNUL(DR))dr = vec3d(1, 0, 0);
		else dr = DR;
	}
	__host__ __device__ void plane::setRaw(vec3d PT, vec3d DR) {
		pt = PT;
		dr = DR;
	}

	__host__ __device__ void plane::setPT(vec3d PT) {
		pt = PT;
	}

	__host__ __device__ bool plane::setDR(vec3d DR) {
		if (vec3d::isNUL(DR)) {
			dr = vec3d(1, 0, 0);
			return true;
		}
		dr = DR;
		return false;
	}

	__host__ __device__ void plane::setDRRaw_s(vec3d DR) {
		if (vec3d::isNUL(DR))dr = vec3d(1, 0, 0);
		else dr = DR;
	}

	__host__ __device__ void plane::setDRRaw(vec3d DR) {
		dr = DR;
	}
	__host__ __device__ vec3d plane::getPt() { return pt; }
	__host__ __device__ vec3d plane::getDr() { return dr; }



	//line functions

	__host__ __device__ bool getPt(line l, vec3d& coord, coordinateName coordGiven) {
		if (coordGiven == coordinateName::xCoordinate) {//coord is x coord
			if (l.getDr().x == 0) {
				coord = (l.getPt());
				return 1;//error
			}
			else {
				coord = vec3d(coord.x, l.getPt().y + l.getDr().y * (coord.x - l.getPt().x) / l.getDr().x, l.getPt().z + l.getDr().z * (coord.x - l.getPt().x) / l.getDr().x);
			}
		}
		else if (coordGiven == coordinateName::yCoordinate) {//coord is y coord
			if (l.getDr().y == 0) {
				coord = (l.getPt());
				return 1;//error
			}
			else {
				coord = vec3d(l.getPt().x + l.getDr().x * (coord.y - l.getPt().y) / l.getDr().y, coord.y, l.getPt().z + l.getDr().z * (coord.y - l.getPt().y) / l.getDr().y);
			}
		}
		else {//z coordinate
			if (l.getDr().z == 0) {
				coord = (l.getPt());
				return 1;//error
			}
			else {
				coord = vec3d(l.getPt().x + l.getDr().x * (coord.z - l.getPt().z) / l.getDr().z, l.getPt().y + l.getDr().y * (coord.z - l.getPt().z) / l.getDr().z, coord.z);
			}
		}
		return 0;//no error
	}

	__host__ __device__ void getPtRaw_s(line l, vec3d& coord, coordinateName coordGiven) {
		if (coordGiven == coordinateName::xCoordinate) {//coord is x coord
			if (l.getDr().x == 0) {
				coord = (l.getPt());
			}
			else {
				coord = vec3d(coord.x, l.getPt().y + l.getDr().y * (coord.x - l.getPt().x) / l.getDr().x, l.getPt().z + l.getDr().z * (coord.x - l.getPt().x) / l.getDr().x);
			}
		}
		else if (coordGiven == coordinateName::yCoordinate) {//coord is y coord
			if (l.getDr().y == 0) {
				coord = (l.getPt());
			}
			else {
				coord = vec3d(l.getPt().x + l.getDr().x * (coord.y - l.getPt().y) / l.getDr().y, coord.y, l.getPt().z + l.getDr().z * (coord.y - l.getPt().y) / l.getDr().y);
			}
		}
		else {//z coordinate
			if (l.getDr().z == 0) {
				coord = (l.getPt());
			}
			else {
				coord = vec3d(l.getPt().x + l.getDr().x * (coord.z - l.getPt().z) / l.getDr().z, l.getPt().y + l.getDr().y * (coord.z - l.getPt().z) / l.getDr().z, coord.z);
			}
		}
	}

	__host__ __device__ void getPtRaw(line l, vec3d& coord, coordinateName coordGiven) {
		if (coordGiven == coordinateName::xCoordinate) {//coord is x coord
				coord = vec3d(coord.x, l.getPt().y + l.getDr().y * (coord.x - l.getPt().x) / l.getDr().x, l.getPt().z + l.getDr().z * (coord.x - l.getPt().x) / l.getDr().x);
		}
		else if (coordGiven == coordinateName::yCoordinate) {//coord is y coord
				coord = vec3d(l.getPt().x + l.getDr().x * (coord.y - l.getPt().y) / l.getDr().y, coord.y, l.getPt().z + l.getDr().z * (coord.y - l.getPt().y) / l.getDr().y);
		}
		else {//z coordinate
			coord = vec3d(l.getPt().x + l.getDr().x * (coord.z - l.getPt().z) / l.getDr().z, l.getPt().y + l.getDr().y * (coord.z - l.getPt().z) / l.getDr().z, coord.z);
		}
	}

	__host__ __device__ char getPtIn(vec3d start, vec3d end, vec3d& coord, coordinateName coordGiven) {
		char rval = 0;//no error
		if (getPt(line(start, vec3d::subtract(end, start)), coord, coordGiven)) {
			rval = 1;// zero / infinite ans
		}
		else
		{
			double ttlDist = vec3d::subtract(start, end).mag2();
			if (!(vec3d::subtract(coord, start).mag2() <= ttlDist) || !(vec3d::subtract(coord, end).mag2() <= ttlDist)) {
				rval = 2; // out of bounds
			}
		}
		return rval;
	}


	//plane functions

	__host__ __device__ bool getPt(plane p, vec3d& coord, coordinateName coordToFind) {
		if (coordToFind == coordinateName::zCoordinate) {
			if (p.getDr().z == 0) {
				return 1;
			}
			else {
				(coord).z = (vec3d::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().y * (coord).y)) / p.getDr().z;
				return 0;
			}
		}
		else if (coordToFind == coordinateName::xCoordinate) {
			if (p.getDr().x == 0) {
				return 1;
			}
			else {
				(coord).x = (vec3d::dot(p.getPt(), p.getDr()) - (p.getDr().z * (coord).z + p.getDr().y * (coord).y)) / p.getDr().x;
				return 0;
			}
		}
		else {
			if (p.getDr().y == 0) {
				return 1;
			}
			else {
				(coord).y = (vec3d::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().z * (coord).z)) / p.getDr().y;
				return 0;
			}
		}
	}

	__host__ __device__ void getPtRaw_s(plane p, vec3d& coord, coordinateName coordToFind) {
		if (coordToFind == coordinateName::zCoordinate) {
			if (p.getDr().z != 0)
				(coord).z = (vec3d::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().y * (coord).y)) / p.getDr().z;
		}
		else if (coordToFind == coordinateName::xCoordinate) {
			if (p.getDr().x != 0)
				(coord).x = (vec3d::dot(p.getPt(), p.getDr()) - (p.getDr().z * (coord).z + p.getDr().y * (coord).y)) / p.getDr().x;
		}
		else {
			if (p.getDr().y == 0)
				(coord).y = (vec3d::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().z * (coord).z)) / p.getDr().y;
		}
	}

	__host__ __device__ void getPtRaw(plane p, vec3d& coord, coordinateName coordToFind) {
		if (coordToFind == coordinateName::zCoordinate)
				(coord).z = (vec3d::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().y * (coord).y)) / p.getDr().z;
		else if (coordToFind == coordinateName::xCoordinate)
				(coord).x = (vec3d::dot(p.getPt(), p.getDr()) - (p.getDr().z * (coord).z + p.getDr().y * (coord).y)) / p.getDr().x;
		else
				(coord).y = (vec3d::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().z * (coord).z)) / p.getDr().y;
	}


	//point to point functions

	__host__ __device__ double distance(vec3d p1, vec3d p2) {
		return vec3d::subtract(p1, p2).mag();
	}


	//point and line functions

	__host__ __device__ double distance(vec3d p, line l) {
		return vec3d::cross(l.getDr(), vec3d::subtract(p, l.getPt())).mag() / l.getDr().mag();
	}


	//point and plane functions

	__host__ __device__ double aDistance(vec3d pt, plane p) { // algebraic distance
		return (vec3d::dot(pt, p.getDr()) - vec3d::dot(p.getDr(), p.getPt())) / p.getDr().mag();
	}

	__host__ __device__ vec3d getMirrorImage(vec3d pt, plane pl) {
		//get component perpendicular to pl
		double Component = vec3d::componentRaw_s(vec3d::subtract(pt, pl.getPt()), pl.getDr());
		vec3d normal = pl.getDr();
		normal.normalize();
		return (vec3d::subtract(pt, vec3d::multiply(normal, 2 * Component)));
	}


	//line and line functions

	__host__ __device__ bool coplanar(line l, line m) {
		if (vec3d::dot(vec3d::subtract(l.getPt(), m.getPt()), vec3d::cross(l.getDr(), m.getDr())) == 0) {
			return 1;
		}
		else {
			return 0;
		}
	}

	__host__ __device__ double distance(line l, line m) {
		if (vec3d::isEqual(l.getDr(), m.getDr())) {
			return (vec3d::cross(vec3d::subtract(l.getPt(), m.getPt()), l.getDr()).mag() / l.getDr().mag());
		}
		else {
			vec3d temp = vec3d::cross(l.getDr(), m.getDr());
			return fabs(vec3d::dot(vec3d::subtract(l.getPt(), m.getPt()), temp) / temp.mag());
		}
	}


	//plane plane functions

	__host__ __device__ double distance(plane p1, plane p2) {
		if (vec3d::isEqual(p1.getDr(), p2.getDr())) {
			return fabs((vec3d::dot(p1.getDr(), p1.getPt()) - vec3d::dot(p2.getDr(), p2.getPt())) / p1.getDr().mag());
		}
		else {
			return 0;
		}
	}


	//line and plane functions

	__host__ __device__ vec3d intersection(line l, plane p, bool* error) {
		if (vec3d::dot(l.getDr(), p.getDr()) == 0) {
			*error = true;//error , no solution of infinite solutions
			return l.getPt();
		}
		else {
			*error = false;//no error
			double lambda;
			lambda = (vec3d::dot(p.getPt(), p.getDr()) - vec3d::dot(p.getDr(), l.getPt())) / vec3d::dot(p.getDr(), l.getDr());
			vec3d rVal = vec3d::add(l.getPt(), vec3d::multiply(l.getDr(), lambda));
			return rVal;
		}
	}

	__host__ __device__ vec3d intersectionRaw_s(line l, plane p) {
		if (vec3d::dot(l.getDr(), p.getDr()) == 0) {
			return l.getPt();
		}
		else {
			double lambda;
			lambda = (vec3d::dot(p.getPt(), p.getDr()) - vec3d::dot(p.getDr(), l.getPt())) / vec3d::dot(p.getDr(), l.getDr());
			vec3d rVal = vec3d::add(l.getPt(), vec3d::multiply(l.getDr(), lambda));
			return rVal;
		}
	}

	__host__ __device__ vec3d intersectionRaw(line l, plane p) {
			double lambda;
			lambda = (vec3d::dot(p.getPt(), p.getDr()) - vec3d::dot(p.getDr(), l.getPt())) / vec3d::dot(p.getDr(), l.getDr());
			vec3d rVal = vec3d::add(l.getPt(), vec3d::multiply(l.getDr(), lambda));
			return rVal;
	}


	//ray cast

	__host__ __device__ bool rayCast(line l, plane p, vec3d& intersection) {
		if (vec3d::dot(l.getDr(), p.getDr()) == 0) {
			intersection = l.getPt();
			return 1;// multiple or no results
		}
		else {
			double lambda;
			lambda = (vec3d::dot(p.getPt(), p.getDr()) - vec3d::dot(p.getDr(), l.getPt())) / vec3d::dot(p.getDr(), l.getDr());
			if (lambda < 0) {
				intersection = l.getPt();
				return 1;//no results
			}
			intersection = vec3d::add(l.getPt(), vec3d::multiply(l.getDr(), lambda));
		}
	}
}



namespace linearMathF {//single precesion


	//line class
	__host__ __device__ line::line() {
		setRaw(vec3f(0, 0, 0), vec3f(1, 0, 0));
	}

	__host__ __device__ line::line(vec3f PT, vec3f DR) {
		setRaw_s(PT, DR);
	}

	__host__ __device__ bool line::set(vec3f PT, vec3f DR) {
		pt = PT;
		if (vec3f::isNUL(DR)) {
			dr = vec3f(1, 0, 0);
			return true;//error
		}
		dr = DR;
		return false;
	}

	__host__ __device__ void line::setRaw_s(vec3f PT, vec3f DR) {
		pt = PT;
		if (vec3f::isNUL(DR)) dr = vec3f(1, 0, 0);
		else dr = DR;
	}

	__host__ __device__ void line::setRaw(vec3f PT, vec3f DR) {
		pt = PT;
		dr = DR;
	}



	__host__ __device__ void line::setPT(vec3f PT) {
		pt = PT;
	}

	__host__ __device__ bool line::setDR(vec3f DR) {
		if (vec3f::isNUL(DR)) {
			dr = vec3f(1, 0, 0);
			return true;//error
		}
		dr = DR;
		return false;
	}

	__host__ __device__ void line::setDRRaw_s(vec3f DR) {
		if (vec3f::isNUL(DR))dr = vec3f(1, 0, 0);
		else dr = DR;
	}

	__host__ __device__ void line::setDRRaw(vec3f DR) {
		dr = DR;
	}

	__host__ __device__ vec3f line::getPt() { return pt; }
	__host__ __device__ vec3f line::getDr() { return dr; }

	//plane class
	__host__ __device__ plane::plane() {
		setRaw(vec3f(0, 0, 0), vec3f(1, 0, 0));
	}

	__host__ __device__ plane::plane(vec3f PT, vec3f DR) {
		setRaw_s(PT, DR);
	}

	__host__ __device__ bool plane::set(vec3f PT, vec3f DR) {
		pt = PT;
		if (vec3f::isNUL(DR)) {
			dr = vec3f(1, 0, 0);
			return true;//error
		}
		dr = DR;
		return false;
	}
	__host__ __device__ void plane::setRaw_s(vec3f PT, vec3f DR) {
		pt = PT;
		if (vec3f::isNUL(DR))dr = vec3f(1, 0, 0);
		else dr = DR;
	}
	__host__ __device__ void plane::setRaw(vec3f PT, vec3f DR) {
		pt = PT;
		dr = DR;
	}

	__host__ __device__ void plane::setPT(vec3f PT) {
		pt = PT;
	}

	__host__ __device__ bool plane::setDR(vec3f DR) {
		if (vec3f::isNUL(DR)) {
			dr = vec3f(1, 0, 0);
			return true;
		}
		dr = DR;
		return false;
	}

	__host__ __device__ void plane::setDRRaw_s(vec3f DR) {
		if (vec3f::isNUL(DR))dr = vec3f(1, 0, 0);
		else dr = DR;
	}

	__host__ __device__ void plane::setDRRaw(vec3f DR) {
		dr = DR;
	}
	__host__ __device__ vec3f plane::getPt() { return pt; }
	__host__ __device__ vec3f plane::getDr() { return dr; }



	//line functions

	__host__ __device__ bool getPt(line l, vec3f& coord, coordinateName coordGiven) {
		if (coordGiven == coordinateName::xCoordinate) {//coord is x coord
			if (l.getDr().x == 0) {
				coord = (l.getPt());
				return 1;//error
			}
			else {
				coord = vec3f(coord.x, l.getPt().y + l.getDr().y * (coord.x - l.getPt().x) / l.getDr().x, l.getPt().z + l.getDr().z * (coord.x - l.getPt().x) / l.getDr().x);
			}
		}
		else if (coordGiven == coordinateName::yCoordinate) {//coord is y coord
			if (l.getDr().y == 0) {
				coord = (l.getPt());
				return 1;//error
			}
			else {
				coord = vec3f(l.getPt().x + l.getDr().x * (coord.y - l.getPt().y) / l.getDr().y, coord.y, l.getPt().z + l.getDr().z * (coord.y - l.getPt().y) / l.getDr().y);
			}
		}
		else {//z coordinate
			if (l.getDr().z == 0) {
				coord = (l.getPt());
				return 1;//error
			}
			else {
				coord = vec3f(l.getPt().x + l.getDr().x * (coord.z - l.getPt().z) / l.getDr().z, l.getPt().y + l.getDr().y * (coord.z - l.getPt().z) / l.getDr().z, coord.z);
			}
		}
		return 0;//no error
	}

	__host__ __device__ void getPtRaw_s(line l, vec3f& coord, coordinateName coordGiven) {
		if (coordGiven == coordinateName::xCoordinate) {//coord is x coord
			if (l.getDr().x == 0) {
				coord = (l.getPt());
			}
			else {
				coord = vec3f(coord.x, l.getPt().y + l.getDr().y * (coord.x - l.getPt().x) / l.getDr().x, l.getPt().z + l.getDr().z * (coord.x - l.getPt().x) / l.getDr().x);
			}
		}
		else if (coordGiven == coordinateName::yCoordinate) {//coord is y coord
			if (l.getDr().y == 0) {
				coord = (l.getPt());
			}
			else {
				coord = vec3f(l.getPt().x + l.getDr().x * (coord.y - l.getPt().y) / l.getDr().y, coord.y, l.getPt().z + l.getDr().z * (coord.y - l.getPt().y) / l.getDr().y);
			}
		}
		else {//z coordinate
			if (l.getDr().z == 0) {
				coord = (l.getPt());
			}
			else {
				coord = vec3f(l.getPt().x + l.getDr().x * (coord.z - l.getPt().z) / l.getDr().z, l.getPt().y + l.getDr().y * (coord.z - l.getPt().z) / l.getDr().z, coord.z);
			}
		}
	}

	__host__ __device__ void getPtRaw(line l, vec3f& coord, coordinateName coordGiven) {
		if (coordGiven == coordinateName::xCoordinate) {//coord is x coord
			coord = vec3f(coord.x, l.getPt().y + l.getDr().y * (coord.x - l.getPt().x) / l.getDr().x, l.getPt().z + l.getDr().z * (coord.x - l.getPt().x) / l.getDr().x);
		}
		else if (coordGiven == coordinateName::yCoordinate) {//coord is y coord
			coord = vec3f(l.getPt().x + l.getDr().x * (coord.y - l.getPt().y) / l.getDr().y, coord.y, l.getPt().z + l.getDr().z * (coord.y - l.getPt().y) / l.getDr().y);
		}
		else {//z coordinate
			coord = vec3f(l.getPt().x + l.getDr().x * (coord.z - l.getPt().z) / l.getDr().z, l.getPt().y + l.getDr().y * (coord.z - l.getPt().z) / l.getDr().z, coord.z);
		}
	}

	__host__ __device__ char getPtIn(vec3f start, vec3f end, vec3f& coord, coordinateName coordGiven) {
		char rval = 0;//no error
		if (getPt(line(start, vec3f::subtract(end, start)), coord, coordGiven)) {
			rval = 1;// zero / infinite ans
		}
		else
		{
			float ttlDist = vec3f::subtract(start, end).mag2();
			if (!(vec3f::subtract(coord, start).mag2() <= ttlDist) || !(vec3f::subtract(coord, end).mag2() <= ttlDist)) {
				rval = 2; // out of bounds
			}
		}
		return rval;
	}


	//plane functions

	__host__ __device__ bool getPt(plane p, vec3f& coord, coordinateName coordToFind) {
		if (coordToFind == coordinateName::zCoordinate) {
			if (p.getDr().z == 0) {
				return 1;
			}
			else {
				(coord).z = (vec3f::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().y * (coord).y)) / p.getDr().z;
				return 0;
			}
		}
		else if (coordToFind == coordinateName::xCoordinate) {
			if (p.getDr().x == 0) {
				return 1;
			}
			else {
				(coord).x = (vec3f::dot(p.getPt(), p.getDr()) - (p.getDr().z * (coord).z + p.getDr().y * (coord).y)) / p.getDr().x;
				return 0;
			}
		}
		else {
			if (p.getDr().y == 0) {
				return 1;
			}
			else {
				(coord).y = (vec3f::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().z * (coord).z)) / p.getDr().y;
				return 0;
			}
		}
	}

	__host__ __device__ void getPtRaw_s(plane p, vec3f& coord, coordinateName coordToFind) {
		if (coordToFind == coordinateName::zCoordinate) {
			if (p.getDr().z != 0)
				(coord).z = (vec3f::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().y * (coord).y)) / p.getDr().z;
		}
		else if (coordToFind == coordinateName::xCoordinate) {
			if (p.getDr().x != 0)
				(coord).x = (vec3f::dot(p.getPt(), p.getDr()) - (p.getDr().z * (coord).z + p.getDr().y * (coord).y)) / p.getDr().x;
		}
		else {
			if (p.getDr().y == 0)
				(coord).y = (vec3f::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().z * (coord).z)) / p.getDr().y;
		}
	}

	__host__ __device__ void getPtRaw(plane p, vec3f& coord, coordinateName coordToFind) {
		if (coordToFind == coordinateName::zCoordinate)
			(coord).z = (vec3f::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().y * (coord).y)) / p.getDr().z;
		else if (coordToFind == coordinateName::xCoordinate)
			(coord).x = (vec3f::dot(p.getPt(), p.getDr()) - (p.getDr().z * (coord).z + p.getDr().y * (coord).y)) / p.getDr().x;
		else
			(coord).y = (vec3f::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().z * (coord).z)) / p.getDr().y;
	}


	//point to point functions

	__host__ __device__ float distance(vec3f p1, vec3f p2) {
		return vec3f::subtract(p1, p2).mag();
	}


	//point and line functions

	__host__ __device__ float distance(vec3f p, line l) {
		return vec3f::cross(l.getDr(), vec3f::subtract(p, l.getPt())).mag() / l.getDr().mag();
	}


	//point and plane functions

	__host__ __device__ float aDistance(vec3f pt, plane p) { // algebraic distance
		return (vec3f::dot(pt, p.getDr()) - vec3f::dot(p.getDr(), p.getPt())) / p.getDr().mag();
	}

	__host__ __device__ vec3f getMirrorImage(vec3f pt, plane pl) {
		//get component perpendicular to pl
		float Component = vec3f::componentRaw_s(vec3f::subtract(pt, pl.getPt()), pl.getDr());
		vec3f normal = pl.getDr();
		normal.normalize();
		return (vec3f::subtract(pt, vec3f::multiply(normal, 2 * Component)));
	}


	//line and line functions

	__host__ __device__ bool coplanar(line l, line m) {
		if (vec3f::dot(vec3f::subtract(l.getPt(), m.getPt()), vec3f::cross(l.getDr(), m.getDr())) == 0) {
			return 1;
		}
		else {
			return 0;
		}
	}

	__host__ __device__ float distance(line l, line m) {
		if (vec3f::isEqual(l.getDr(), m.getDr())) {
			return (vec3f::cross(vec3f::subtract(l.getPt(), m.getPt()), l.getDr()).mag() / l.getDr().mag());
		}
		else {
			vec3f temp = vec3f::cross(l.getDr(), m.getDr());
			return fabs(vec3f::dot(vec3f::subtract(l.getPt(), m.getPt()), temp) / temp.mag());
		}
	}


	//plane plane functions

	__host__ __device__ float distance(plane p1, plane p2) {
		if (vec3f::isEqual(p1.getDr(), p2.getDr())) {
			return fabs((vec3f::dot(p1.getDr(), p1.getPt()) - vec3f::dot(p2.getDr(), p2.getPt())) / p1.getDr().mag());
		}
		else {
			return 0;
		}
	}


	//line and plane functions

	__host__ __device__ vec3f intersection(line l, plane p, bool* error) {
		if (vec3f::dot(l.getDr(), p.getDr()) == 0) {
			*error = true;//error , no solution of infinite solutions
			return l.getPt();
		}
		else {
			*error = false;//no error
			float lambda;
			lambda = (vec3f::dot(p.getPt(), p.getDr()) - vec3f::dot(p.getDr(), l.getPt())) / vec3f::dot(p.getDr(), l.getDr());
			vec3f rVal = vec3f::add(l.getPt(), vec3f::multiply(l.getDr(), lambda));
			return rVal;
		}
	}

	__host__ __device__ vec3f intersectionRaw_s(line l, plane p) {
		if (vec3f::dot(l.getDr(), p.getDr()) == 0) {
			return l.getPt();
		}
		else {
			float lambda;
			lambda = (vec3f::dot(p.getPt(), p.getDr()) - vec3f::dot(p.getDr(), l.getPt())) / vec3f::dot(p.getDr(), l.getDr());
			vec3f rVal = vec3f::add(l.getPt(), vec3f::multiply(l.getDr(), lambda));
			return rVal;
		}
	}

	__host__ __device__ vec3f intersectionRaw(line l, plane p) {
		float lambda;
		lambda = (vec3f::dot(p.getPt(), p.getDr()) - vec3f::dot(p.getDr(), l.getPt())) / vec3f::dot(p.getDr(), l.getDr());
		vec3f rVal = vec3f::add(l.getPt(), vec3f::multiply(l.getDr(), lambda));
		return rVal;
	}


	//ray cast

	__host__ __device__ bool rayCast(line l, plane p, vec3f& intersection) {
		if (vec3f::dot(l.getDr(), p.getDr()) == 0) {
			intersection = l.getPt();
			return 1;// multiple or no results
		}
		else {
			float lambda;
			lambda = (vec3f::dot(p.getPt(), p.getDr()) - vec3f::dot(p.getDr(), l.getPt())) / vec3f::dot(p.getDr(), l.getDr());
			if (lambda < 0) {
				intersection = l.getPt();
				return 1;//no results
			}
			intersection = vec3f::add(l.getPt(), vec3f::multiply(l.getDr(), lambda));
		}
	}
}



namespace linearMathLD {//long double precesion


	//line class
	__host__ __device__ line::line() {
		setRaw(vec3ld(0, 0, 0), vec3ld(1, 0, 0));
	}

	__host__ __device__ line::line(vec3ld PT, vec3ld DR) {
		setRaw_s(PT, DR);
	}

	__host__ __device__ bool line::set(vec3ld PT, vec3ld DR) {
		pt = PT;
		if (vec3ld::isNUL(DR)) {
			dr = vec3ld(1, 0, 0);
			return true;//error
		}
		dr = DR;
		return false;
	}

	__host__ __device__ void line::setRaw_s(vec3ld PT, vec3ld DR) {
		pt = PT;
		if (vec3ld::isNUL(DR)) dr = vec3ld(1, 0, 0);
		else dr = DR;
	}

	__host__ __device__ void line::setRaw(vec3ld PT, vec3ld DR) {
		pt = PT;
		dr = DR;
	}



	__host__ __device__ void line::setPT(vec3ld PT) {
		pt = PT;
	}

	__host__ __device__ bool line::setDR(vec3ld DR) {
		if (vec3ld::isNUL(DR)) {
			dr = vec3ld(1, 0, 0);
			return true;//error
		}
		dr = DR;
		return false;
	}

	__host__ __device__ void line::setDRRaw_s(vec3ld DR) {
		if (vec3ld::isNUL(DR))dr = vec3ld(1, 0, 0);
		else dr = DR;
	}

	__host__ __device__ void line::setDRRaw(vec3ld DR) {
		dr = DR;
	}

	__host__ __device__ vec3ld line::getPt() { return pt; }
	__host__ __device__ vec3ld line::getDr() { return dr; }

	//plane class
	__host__ __device__ plane::plane() {
		setRaw(vec3ld(0, 0, 0), vec3ld(1, 0, 0));
	}

	__host__ __device__ plane::plane(vec3ld PT, vec3ld DR) {
		setRaw_s(PT, DR);
	}

	__host__ __device__ bool plane::set(vec3ld PT, vec3ld DR) {
		pt = PT;
		if (vec3ld::isNUL(DR)) {
			dr = vec3ld(1, 0, 0);
			return true;//error
		}
		dr = DR;
		return false;
	}
	__host__ __device__ void plane::setRaw_s(vec3ld PT, vec3ld DR) {
		pt = PT;
		if (vec3ld::isNUL(DR))dr = vec3ld(1, 0, 0);
		else dr = DR;
	}
	__host__ __device__ void plane::setRaw(vec3ld PT, vec3ld DR) {
		pt = PT;
		dr = DR;
	}

	__host__ __device__ void plane::setPT(vec3ld PT) {
		pt = PT;
	}

	__host__ __device__ bool plane::setDR(vec3ld DR) {
		if (vec3ld::isNUL(DR)) {
			dr = vec3ld(1, 0, 0);
			return true;
		}
		dr = DR;
		return false;
	}

	__host__ __device__ void plane::setDRRaw_s(vec3ld DR) {
		if (vec3ld::isNUL(DR))dr = vec3ld(1, 0, 0);
		else dr = DR;
	}

	__host__ __device__ void plane::setDRRaw(vec3ld DR) {
		dr = DR;
	}
	__host__ __device__ vec3ld plane::getPt() { return pt; }
	__host__ __device__ vec3ld plane::getDr() { return dr; }



	//line functions

	__host__ __device__ bool getPt(line l, vec3ld& coord, coordinateName coordGiven) {
		if (coordGiven == coordinateName::xCoordinate) {//coord is x coord
			if (l.getDr().x == 0) {
				coord = (l.getPt());
				return 1;//error
			}
			else {
				coord = vec3ld(coord.x, l.getPt().y + l.getDr().y * (coord.x - l.getPt().x) / l.getDr().x, l.getPt().z + l.getDr().z * (coord.x - l.getPt().x) / l.getDr().x);
			}
		}
		else if (coordGiven == coordinateName::yCoordinate) {//coord is y coord
			if (l.getDr().y == 0) {
				coord = (l.getPt());
				return 1;//error
			}
			else {
				coord = vec3ld(l.getPt().x + l.getDr().x * (coord.y - l.getPt().y) / l.getDr().y, coord.y, l.getPt().z + l.getDr().z * (coord.y - l.getPt().y) / l.getDr().y);
			}
		}
		else {//z coordinate
			if (l.getDr().z == 0) {
				coord = (l.getPt());
				return 1;//error
			}
			else {
				coord = vec3ld(l.getPt().x + l.getDr().x * (coord.z - l.getPt().z) / l.getDr().z, l.getPt().y + l.getDr().y * (coord.z - l.getPt().z) / l.getDr().z, coord.z);
			}
		}
		return 0;//no error
	}

	__host__ __device__ void getPtRaw_s(line l, vec3ld& coord, coordinateName coordGiven) {
		if (coordGiven == coordinateName::xCoordinate) {//coord is x coord
			if (l.getDr().x == 0) {
				coord = (l.getPt());
			}
			else {
				coord = vec3ld(coord.x, l.getPt().y + l.getDr().y * (coord.x - l.getPt().x) / l.getDr().x, l.getPt().z + l.getDr().z * (coord.x - l.getPt().x) / l.getDr().x);
			}
		}
		else if (coordGiven == coordinateName::yCoordinate) {//coord is y coord
			if (l.getDr().y == 0) {
				coord = (l.getPt());
			}
			else {
				coord = vec3ld(l.getPt().x + l.getDr().x * (coord.y - l.getPt().y) / l.getDr().y, coord.y, l.getPt().z + l.getDr().z * (coord.y - l.getPt().y) / l.getDr().y);
			}
		}
		else {//z coordinate
			if (l.getDr().z == 0) {
				coord = (l.getPt());
			}
			else {
				coord = vec3ld(l.getPt().x + l.getDr().x * (coord.z - l.getPt().z) / l.getDr().z, l.getPt().y + l.getDr().y * (coord.z - l.getPt().z) / l.getDr().z, coord.z);
			}
		}
	}

	__host__ __device__ void getPtRaw(line l, vec3ld& coord, coordinateName coordGiven) {
		if (coordGiven == coordinateName::xCoordinate) {//coord is x coord
			coord = vec3ld(coord.x, l.getPt().y + l.getDr().y * (coord.x - l.getPt().x) / l.getDr().x, l.getPt().z + l.getDr().z * (coord.x - l.getPt().x) / l.getDr().x);
		}
		else if (coordGiven == coordinateName::yCoordinate) {//coord is y coord
			coord = vec3ld(l.getPt().x + l.getDr().x * (coord.y - l.getPt().y) / l.getDr().y, coord.y, l.getPt().z + l.getDr().z * (coord.y - l.getPt().y) / l.getDr().y);
		}
		else {//z coordinate
			coord = vec3ld(l.getPt().x + l.getDr().x * (coord.z - l.getPt().z) / l.getDr().z, l.getPt().y + l.getDr().y * (coord.z - l.getPt().z) / l.getDr().z, coord.z);
		}
	}

	__host__ __device__ char getPtIn(vec3ld start, vec3ld end, vec3ld& coord, coordinateName coordGiven) {
		char rval = 0;//no error
		if (getPt(line(start, vec3ld::subtract(end, start)), coord, coordGiven)) {
			rval = 1;// zero / infinite ans
		}
		else
		{
			long double ttlDist = vec3ld::subtract(start, end).mag2();
			if (!(vec3ld::subtract(coord, start).mag2() <= ttlDist) || !(vec3ld::subtract(coord, end).mag2() <= ttlDist)) {
				rval = 2; // out of bounds
			}
		}
		return rval;
	}


	//plane functions

	__host__ __device__ bool getPt(plane p, vec3ld& coord, coordinateName coordToFind) {
		if (coordToFind == coordinateName::zCoordinate) {
			if (p.getDr().z == 0) {
				return 1;
			}
			else {
				(coord).z = (vec3ld::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().y * (coord).y)) / p.getDr().z;
				return 0;
			}
		}
		else if (coordToFind == coordinateName::xCoordinate) {
			if (p.getDr().x == 0) {
				return 1;
			}
			else {
				(coord).x = (vec3ld::dot(p.getPt(), p.getDr()) - (p.getDr().z * (coord).z + p.getDr().y * (coord).y)) / p.getDr().x;
				return 0;
			}
		}
		else {
			if (p.getDr().y == 0) {
				return 1;
			}
			else {
				(coord).y = (vec3ld::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().z * (coord).z)) / p.getDr().y;
				return 0;
			}
		}
	}

	__host__ __device__ void getPtRaw_s(plane p, vec3ld& coord, coordinateName coordToFind) {
		if (coordToFind == coordinateName::zCoordinate) {
			if (p.getDr().z != 0)
				(coord).z = (vec3ld::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().y * (coord).y)) / p.getDr().z;
		}
		else if (coordToFind == coordinateName::xCoordinate) {
			if (p.getDr().x != 0)
				(coord).x = (vec3ld::dot(p.getPt(), p.getDr()) - (p.getDr().z * (coord).z + p.getDr().y * (coord).y)) / p.getDr().x;
		}
		else {
			if (p.getDr().y == 0)
				(coord).y = (vec3ld::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().z * (coord).z)) / p.getDr().y;
		}
	}

	__host__ __device__ void getPtRaw(plane p, vec3ld& coord, coordinateName coordToFind) {
		if (coordToFind == coordinateName::zCoordinate)
			(coord).z = (vec3ld::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().y * (coord).y)) / p.getDr().z;
		else if (coordToFind == coordinateName::xCoordinate)
			(coord).x = (vec3ld::dot(p.getPt(), p.getDr()) - (p.getDr().z * (coord).z + p.getDr().y * (coord).y)) / p.getDr().x;
		else
			(coord).y = (vec3ld::dot(p.getPt(), p.getDr()) - (p.getDr().x * (coord).x + p.getDr().z * (coord).z)) / p.getDr().y;
	}


	//point to point functions

	__host__ __device__ long double distance(vec3ld p1, vec3ld p2) {
		return vec3ld::subtract(p1, p2).mag();
	}


	//point and line functions

	__host__ __device__ long double distance(vec3ld p, line l) {
		return vec3ld::cross(l.getDr(), vec3ld::subtract(p, l.getPt())).mag() / l.getDr().mag();
	}


	//point and plane functions

	__host__ __device__ long double aDistance(vec3ld pt, plane p) { // algebraic distance
		return (vec3ld::dot(pt, p.getDr()) - vec3ld::dot(p.getDr(), p.getPt())) / p.getDr().mag();
	}

	__host__ __device__ vec3ld getMirrorImage(vec3ld pt, plane pl) {
		//get component perpendicular to pl
		long double Component = vec3ld::componentRaw_s(vec3ld::subtract(pt, pl.getPt()), pl.getDr());
		vec3ld normal = pl.getDr();
		normal.normalize();
		return (vec3ld::subtract(pt, vec3ld::multiply(normal, 2 * Component)));
	}


	//line and line functions

	__host__ __device__ bool coplanar(line l, line m) {
		if (vec3ld::dot(vec3ld::subtract(l.getPt(), m.getPt()), vec3ld::cross(l.getDr(), m.getDr())) == 0) {
			return 1;
		}
		else {
			return 0;
		}
	}

	__host__ __device__ long double distance(line l, line m) {
		if (vec3ld::isEqual(l.getDr(), m.getDr())) {
			return (vec3ld::cross(vec3ld::subtract(l.getPt(), m.getPt()), l.getDr()).mag() / l.getDr().mag());
		}
		else {
			vec3ld temp = vec3ld::cross(l.getDr(), m.getDr());
			return fabs(vec3ld::dot(vec3ld::subtract(l.getPt(), m.getPt()), temp) / temp.mag());
		}
	}


	//plane plane functions

	__host__ __device__ long double distance(plane p1, plane p2) {
		if (vec3ld::isEqual(p1.getDr(), p2.getDr())) {
			return fabs((vec3ld::dot(p1.getDr(), p1.getPt()) - vec3ld::dot(p2.getDr(), p2.getPt())) / p1.getDr().mag());
		}
		else {
			return 0;
		}
	}


	//line and plane functions

	__host__ __device__ vec3ld intersection(line l, plane p, bool* error) {
		if (vec3ld::dot(l.getDr(), p.getDr()) == 0) {
			*error = true;//error , no solution of infinite solutions
			return l.getPt();
		}
		else {
			*error = false;//no error
			long double lambda;
			lambda = (vec3ld::dot(p.getPt(), p.getDr()) - vec3ld::dot(p.getDr(), l.getPt())) / vec3ld::dot(p.getDr(), l.getDr());
			vec3ld rVal = vec3ld::add(l.getPt(), vec3ld::multiply(l.getDr(), lambda));
			return rVal;
		}
	}

	__host__ __device__ vec3ld intersectionRaw_s(line l, plane p) {
		if (vec3ld::dot(l.getDr(), p.getDr()) == 0) {
			return l.getPt();
		}
		else {
			long double lambda;
			lambda = (vec3ld::dot(p.getPt(), p.getDr()) - vec3ld::dot(p.getDr(), l.getPt())) / vec3ld::dot(p.getDr(), l.getDr());
			vec3ld rVal = vec3ld::add(l.getPt(), vec3ld::multiply(l.getDr(), lambda));
			return rVal;
		}
	}

	__host__ __device__ vec3ld intersectionRaw(line l, plane p) {
		long double lambda;
		lambda = (vec3ld::dot(p.getPt(), p.getDr()) - vec3ld::dot(p.getDr(), l.getPt())) / vec3ld::dot(p.getDr(), l.getDr());
		vec3ld rVal = vec3ld::add(l.getPt(), vec3ld::multiply(l.getDr(), lambda));
		return rVal;
	}


	//ray cast

	__host__ __device__ bool rayCast(line l, plane p, vec3ld& intersection) {
		if (vec3ld::dot(l.getDr(), p.getDr()) == 0) {
			intersection = l.getPt();
			return 1;// multiple or no results
		}
		else {
			long double lambda;
			lambda = (vec3ld::dot(p.getPt(), p.getDr()) - vec3ld::dot(p.getDr(), l.getPt())) / vec3ld::dot(p.getDr(), l.getDr());
			if (lambda < 0) {
				intersection = l.getPt();
				return 1;//no results
			}
			intersection = vec3ld::add(l.getPt(), vec3ld::multiply(l.getDr(), lambda));
		}
	}
}