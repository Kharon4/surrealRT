#include "hip/hip_runtime.h"
#include "rendering.cuh"

#include <thread>

#ifdef __GPUDEBUG
#include <iostream>
#endif

#define threadNo 1024
#define blockNo(Threads) ((Threads/threadNo) + 1)

__global__
void initRays(short xRes , short yRes , vec3f vertex , vec3f topLeft , vec3f right , vec3f down , linearMath::linef * rays) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= (xRes * yRes))return;
	
	short x, short y;
	x = tId % xRes;
	y = tId / xRes;

	rays[tId].setRaw_s(vertex, vec3f::subtract(vec3f::add(topLeft, vec3f::add(vec3f::multiply(right, (x + 0.5) / xRes), vec3f::multiply(down, (y + 0.5) / yRes))), vertex));
}

__device__ __host__ void calculateMeshConstraints(mesh* Mesh , meshConstrained *meshC){
	vec3f plNormal = vec3f::cross(Mesh->pts[1] - Mesh->pts[0], Mesh->pts[2] - Mesh->pts[0]);
	meshC->planeNormal = vec3f::normalizeRaw_s(plNormal);
	meshC->sidePlaneNormals[0] = vec3f::normalizeRaw_s(vec3f::cross(plNormal, Mesh->pts[1] - Mesh->pts[0]));
	meshC->sidePlaneNormals[1] = vec3f::normalizeRaw_s(vec3f::cross(plNormal, Mesh->pts[2] - Mesh->pts[1]));
	meshC->sidePlaneNormals[2] = vec3f::normalizeRaw_s(vec3f::cross(plNormal, Mesh->pts[0] - Mesh->pts[2]));
}

__global__
void initMesh(meshShaded* Mesh, meshConstrained* meshC, size_t noOfThreads) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= noOfThreads)return;
	calculateMeshConstraints(&((Mesh + tId)->M), meshC + tId);
}

__device__ __host__
void getClosestIntersection(meshShaded * Mesh ,meshConstrained* meshC, linearMath::linef ray, size_t noTrs,  meshShaded * &OUTmesh, meshConstrained * &OUTMeshC , float& OUTlambda, vec3f& OUTpt) {
	OUTmesh = nullptr;
	OUTMeshC = nullptr;
	OUTlambda = -1;
	double tempDist;
	for (size_t i = 0; i < noTrs; ++i) {
		
		switch (Mesh[i].colShader->meshVProp)
		{
		case (meshVisibilityProperties::inActive):
			tempDist = -1;
			break;
		case (meshVisibilityProperties::frontActive):
			{	
				float dotCalculated = vec3f::dot(ray.getDr(), meshC[i].planeNormal);
				if (dotCalculated > 0) {
					tempDist = vec3f::dot(Mesh[i].M.pts[0] - ray.getPt(), meshC[i].planeNormal) / dotCalculated;
				}
				else tempDist = -1;
			}
			break;

		case (meshVisibilityProperties::backActive):
			{
				float dotCalculated = vec3f::dot(ray.getDr(), meshC[i].planeNormal);
				if (dotCalculated < 0) {
					tempDist = vec3f::dot(Mesh[i].M.pts[0] - ray.getPt(), meshC[i].planeNormal) / dotCalculated;
				}
				else tempDist = -1;
			}
				break;
		case (meshVisibilityProperties::frontBackActive):
			{
				float dotCalculated = vec3f::dot(ray.getDr(), meshC[i].planeNormal);
				if (dotCalculated != 0) {
					tempDist = vec3f::dot(Mesh[i].M.pts[0] - ray.getPt(), meshC[i].planeNormal) / dotCalculated;
				}
				else tempDist = -1;
			}
			break;

		}


		//check for visibility
		if (tempDist > 0 && (tempDist < OUTlambda || OUTlambda < 0)) {
			//check for inside
			vec3f pt = linearMath::getPt(ray, tempDist);
			if (vec3f::dot(pt - Mesh[i].M.pts[0], meshC[i].sidePlaneNormals[0]) < 0)continue;
			if (vec3f::dot(pt - Mesh[i].M.pts[1], meshC[i].sidePlaneNormals[1]) < 0)continue;
			if (vec3f::dot(pt - Mesh[i].M.pts[2], meshC[i].sidePlaneNormals[2]) < 0)continue;
			//inside
			OUTlambda = tempDist;

			OUTpt = pt;
			OUTmesh = Mesh + i;
			OUTMeshC = meshC + i;
		}
	}
}

__global__
void getIntersections(linearMath::linef* rays, size_t noRays, meshShaded* trs, meshConstrained* collTrs, size_t noTrs, color* displayData, chromaticShader* defaultShader) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= noRays)return;

	fragmentProperties fp;
	fp.ray = rays + tId;
	meshShaded* outM;
	getClosestIntersection(trs, collTrs, *fp.ray, noTrs, outM, fp.ip.MC, fp.ip.lambda, fp.ip.pt);

	//shade
	if (outM == nullptr) {
		fp.ip.M = nullptr;
		displayData[tId] = (defaultShader)->shade(fp);
	}
	else {
		fp.ip.M = &(outM->M);
		displayData[tId] = outM->colShader->shade(fp);
	}
}


__global__
void getByteColor(color* data, colorBYTE* dataByte, float min, float delta, size_t noThreads) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= noThreads)return;
	color rval = data[tId];
	rval -= vec3f(min, min, min);
	rval *= 256 / delta;
	if (rval.x > 255)dataByte[tId].r = 255;
	else if (rval.x < 0)dataByte[tId].r = 0;
	else dataByte[tId].r = (unsigned char)rval.x;
	if (rval.y > 255)dataByte[tId].g = 255;
	else if (rval.y < 0)dataByte[tId].g = 0;
	else dataByte[tId].g = (unsigned char)rval.y;
	if (rval.z > 255)dataByte[tId].b = 255;
	else if (rval.z < 0)dataByte[tId].b = 0;
	else dataByte[tId].b = (unsigned char)rval.z;
}

void displayCudaError(size_t id = 0) {
#ifdef __GPUDEBUG
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	std::cout << "#" << id <<"  "<< hipGetErrorName(err)<< std::endl;
	if (err != hipError_t::hipSuccess) {
		int x;
		std::cin >> x;
	}
#else

#endif
}

void generateGPUDisplatData(colorBYTE** data , camera cam) {
	displayCudaError(11);
	hipMalloc(data, sizeof(colorBYTE) * cam.sc.resX * cam.sc.resY);
	displayCudaError(10);
}

void renderIntermediate(camera cam,colorBYTE* DataByte, meshShaded* meshS, meshConstrained* meshC, size_t noTrs) {
	displayCudaError(9);
	linearMath::linef* rays;
	displayCudaError(8);
	hipMalloc(&rays, sizeof(linearMath::linef) * cam.sc.resX * cam.sc.resY);
	displayCudaError(7);
	initRays << <blockNo(cam.sc.resX * cam.sc.resY), threadNo >> > (cam.sc.resX, cam.sc.resY, cam.vertex, cam.sc.screenCenter - cam.sc.halfRight + cam.sc.halfUp, cam.sc.halfRight * 2, cam.sc.halfUp * -2, rays);
	displayCudaError(1);
	skyboxCPU defaultShader(color(0, 0, 128), color(-200, -200, -200), color(150, 0, 0), color(0, 0, 64), color(0, 0, 64), color(0, 0, 64));
	//solidColCPU defaultShader(color(0, 0, 0));
	displayCudaError(2);
	color* Data;
	hipMalloc(&Data, sizeof(color) * cam.sc.resX * cam.sc.resY);
	displayCudaError(3);
	getIntersections << <blockNo(cam.sc.resX * cam.sc.resY), threadNo >> > (rays, cam.sc.resX * cam.sc.resY, meshS, meshC, noTrs, Data, defaultShader.getGPUPtr());
	displayCudaError(4);
	getByteColor << <blockNo(cam.sc.resX * cam.sc.resY), threadNo >> > (Data, DataByte, 0, 256, cam.sc.resX * cam.sc.resY);
	displayCudaError(5);
	hipFree(Data);
	hipFree(rays);
	displayCudaError(6);
}

void cpyData(colorBYTE* data , BYTE * displayData, camera cam) {
	hipDeviceSynchronize();
	displayCudaError(12);
	hipMemcpy(displayData, data, sizeof(colorBYTE) * cam.sc.resX * cam.sc.resY, hipMemcpyKind::hipMemcpyDeviceToHost);
	displayCudaError(13);
	hipFree(data);
	displayCudaError(14);
}

void Render(camera cam,BYTE *data, meshShaded * meshS , meshConstrained * meshC , size_t noTrs) {
	colorBYTE* displayData;
	generateGPUDisplatData(&displayData, cam);
	renderIntermediate(cam, displayData, meshS, meshC, noTrs);
	cpyData(displayData, data, cam);
}


void graphicalWorld::render(camera cam, BYTE* data) {


	bool updated=false;
	meshShaded* devPtr = meshS->getDevice(&updated);
	if(updated){
		initMesh<<<blockNo(meshS->getNoElements()),threadNo>>>(devPtr, meshC->getDevice(), meshS->getNoElements());
	}
	Render(cam, data, meshS->getDevice(), meshC->getDevice(), meshS->getNoElements());

}

void graphicalWorld::render(camera cam, BYTE* data, std::function<void()> drawCall) {
	std::thread draw(drawCall);
	renderPartial(cam);
	draw.join();
	copyData(cam, data);
}

void graphicalWorld::renderPartial(camera cam) {
	
	bool updated = false;
	meshShaded* devPtr = meshS->getDevice(&updated);
	if (updated) {
		initMesh << <blockNo(meshS->getNoElements()), threadNo >> > (devPtr, meshC->getDevice(), meshS->getNoElements());
	}
	if (tempData != nullptr) {
		hipDeviceSynchronize();
		hipFree(tempData);
		tempData = nullptr;
		displayCudaError(16);
	}
	generateGPUDisplatData(&tempData, cam);
	renderIntermediate(cam, tempData, meshS->getDevice(), meshC->getDevice(), meshS->getNoElements());
	//Render(cam, data, meshS->getDevice(), meshC->getDevice(), meshS->getNoElements());
}

void graphicalWorld::copyData(camera cam, BYTE* data) {
	if (tempData != nullptr) {
		cpyData(tempData, data, cam);
		tempData = nullptr;
	}
}