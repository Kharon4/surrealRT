#include "hip/hip_runtime.h"
#include "rendering.cuh"
#include <iostream>

#define threadNo 1024
#define blockNo(Threads) Threads/threadNo


__global__
void initRays(short xRes , short yRes , vec3d vertex , vec3d topLeft , vec3d right , vec3d down , linearMathD::line * rays) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= (xRes * yRes))return;
	
	short x, short y;
	x = tId % xRes;
	y = tId / yRes;

	rays[tId].setRaw_s(vertex, vec3d::subtract(vec3d::add(topLeft, vec3d::add(vec3d::multiply(right, (x + 0.5) / xRes), vec3d::multiply(down, (y + 0.5) / yRes))), vertex));
}

__global__
void getIntersections(linearMathD::line * rays , mesh ** intersections , size_t noRays) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= noRays)return;
	intersections[tId] = nullptr;
}

__global__
void shadeKernel(mesh** interactions, color* data, chromaticShader** defaultShader , size_t maxNo) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= maxNo)return;
	shaderData df;
	if (interactions[tId] == nullptr)data[tId] = (*defaultShader)->shade(df);
}

__global__
void createShader(chromaticShader ** ptr){
	color c;
	c.r = 0;
	c.g = 100;
	c.b = 200;
	*ptr = new solidColor(c);
	//ptr->c.r = 0;
	//ptr->c.g = 100;
	//ptr->c.b = 200;
}

__global__
void deleteShader(chromaticShader** ptr)
{
	delete (*ptr);
}


void render(camera cam,BYTE *data) {
	linearMathD::line * rays;
	hipMalloc(&rays, sizeof(linearMathD::line) * cam.sc.resX * cam.sc.resY);
	//std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
	initRays<<<threadNo , blockNo(cam.sc.resX*cam.sc.resY)>>>(cam.sc.resX, cam.sc.resY, cam.vertex, cam.sc.screenCenter - cam.sc.halfRight + cam.sc.halfUp, cam.sc.halfRight * 2, cam.sc.halfUp * -2, rays);
	//std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
	mesh** intersections;
	hipMalloc(&intersections, sizeof(mesh*) * cam.sc.resX * cam.sc.resY);
	//std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
	getIntersections << <threadNo, blockNo(cam.sc.resX * cam.sc.resY) >> > (rays, intersections, cam.sc.resX * cam.sc.resY);
	//std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
	chromaticShader** sc;
	hipMalloc(&sc, sizeof(chromaticShader*));
	createShader<<<1,1>>>(sc);
	color* Data;
	hipMalloc(&Data, sizeof(color) * cam.sc.resX * cam.sc.resY);
	hipDeviceSynchronize();
	std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
	shadeKernel << <threadNo, blockNo(cam.sc.resX * cam.sc.resY) >> > (intersections, Data, sc, cam.sc.resX * cam.sc.resY);

	hipDeviceSynchronize();
	std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
	deleteShader<<<1,1>>>(sc);
	hipDeviceSynchronize();
	std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
	hipDeviceSynchronize();
	std::cout << hipGetErrorName(hipMemcpy(data, Data, sizeof(color) * cam.sc.resX * cam.sc.resY, hipMemcpyKind::hipMemcpyDeviceToHost))<<std::endl;
	hipDeviceSynchronize();
	hipFree(Data);
	hipFree(sc);
	hipFree(intersections);
	hipFree(rays);
}