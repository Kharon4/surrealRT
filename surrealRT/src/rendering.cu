#include "hip/hip_runtime.h"
#include "rendering.cuh"

#include <thread>


#ifdef __GPUDEBUG
#include <iostream>
#endif

#define threadNo 1024
#define blockNo(Threads) ((Threads/threadNo) + 1)

__global__
void initRays(short xRes , short yRes , vec3f vertex , vec3f topLeft , vec3f right , vec3f down , linearMath::linef * rays) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= (xRes * yRes))return;
	
	short x, short y;
	x = tId % xRes;
	y = tId / xRes;

	rays[tId].setRaw_s(vertex, vec3f::subtract(vec3f::add(topLeft, vec3f::add(vec3f::multiply(right, (x + 0.5) / xRes), vec3f::multiply(down, (y + 0.5) / yRes))), vertex));
}

__device__ __host__ void calculateMeshConstraints(mesh* Mesh , meshConstrained *meshC){
	meshC->a = Mesh->pts[1] - Mesh->pts[0];
	vec3f b = Mesh->pts[2] - Mesh->pts[0];
	vec3f plNormal = vec3f::cross(meshC->a, b);
	meshC->planeNormal = vec3f::normalizeRaw_s(plNormal);
	meshC->sn = vec3f::normalizeRaw_s(vec3f::cross(meshC->planeNormal, meshC->a));

	meshC->coordCalcData.x = vec3f::dot(meshC->sn, b);
	meshC->coordCalcData.y = vec3f::dot(b, meshC->a);
	meshC->coordCalcData.z = meshC->a.mag2();

	if (meshC->coordCalcData.x == 0 || meshC->coordCalcData.z == 0) {
		//do nothing
	}
	else {
		meshC->coordCalcData.x = 1 / meshC->coordCalcData.x;
		meshC->coordCalcData.z = 1 / meshC->coordCalcData.z;
	}

}

__global__
void initMesh(meshShaded* Mesh, meshConstrained* meshC, size_t noOfThreads) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= noOfThreads)return;
	calculateMeshConstraints(&((Mesh + tId)->M), meshC + tId);
}

__device__ __host__
void getClosestIntersection(meshShaded * Mesh ,meshConstrained* meshC, size_t noTrs, fragmentProperties &fp) {
	fp.ip.M = nullptr;
	fp.ip.MC = nullptr;
	fp.ip.lambda = -1;
	fp.ip.trId = UINT_MAX;
	double tempDist;
	linearMath::linef ray = (*fp.ray);
	for (size_t i = 0; i < noTrs; ++i) {
		
		if (Mesh[i].colShader->meshVProp == meshVisibilityProperties::inActive) {
			tempDist = -1;
		}
		else {
			bool calc = false;
			float dotCalculated = vec3f::dot(ray.getDr(), meshC[i].planeNormal);
			if (Mesh[i].colShader->meshVProp == meshVisibilityProperties::frontBackActive) {
				if (dotCalculated != 0)calc = true;
			}
			else {
				if (dotCalculated * (signed char)Mesh[i].colShader->meshVProp < 0)calc = true;
			}

			if (calc) {
				tempDist = vec3f::dot(Mesh[i].M.pts[0] - ray.getPt(), meshC[i].planeNormal) / dotCalculated;
			}
			else {
				tempDist = -1;
			}
		}
		


		//check for visibility
		if (tempDist > 0 && (tempDist < fp.ip.lambda || fp.ip.lambda < 0)) {
			//check for inside
			vec3f pt = linearMath::getPt(ray, tempDist);
			vec3f v = pt - Mesh[i].M.pts[0];
			float l1, l2;
			l2 = vec3f::dot(v, meshC[i].sn) * meshC[i].coordCalcData.x;
			l1 = (vec3f::dot(v, meshC[i].a) - l2 * meshC[i].coordCalcData.y) * meshC[i].coordCalcData.z;
			
			//inside
			if (!(l1 > 0))continue;
			if (!(l2 > 0))continue;
			if ((l1+l2 > 1))continue;

			//write data
			fp.ip.lambda = tempDist;
			fp.ip.pt = pt;
			fp.ip.trId = i;
			fp.ip.cx = l1;
			fp.ip.cy = l2;
		}
	}

	if (fp.ip.trId != UINT_MAX) {
		fp.ip.M = &(Mesh[fp.ip.trId].M);
		fp.ip.MC = meshC + fp.ip.trId;
	}
}


__device__ 
inline void getIntersectionsInternal(linearMath::linef* ray, meshShaded* trs, meshConstrained* collTrs, size_t noTrs, color* pixelData, chromaticShader* defaultShader) {
	fragmentProperties fp;
	fp.ray = ray;
	getClosestIntersection(trs, collTrs, noTrs, fp);

	//shade
	if (fp.ip.trId == UINT_MAX) {
		*pixelData = (defaultShader)->shade(fp);
	}
	else {
		*pixelData = trs[fp.ip.trId].colShader->shade(fp);
	}
}

__global__
void getIntersections(linearMath::linef* rays, size_t noRays, meshShaded* trs, meshConstrained* collTrs, size_t noTrs, color* displayData, chromaticShader* defaultShader) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= noRays)return;

	getIntersectionsInternal(rays + tId, trs, collTrs, noTrs, displayData + tId, defaultShader);
}


__global__
void getByteColor(color* data, colorBYTE* dataByte, float min, float delta, size_t noThreads) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= noThreads)return;
	color rval = data[tId];
	rval -= vec3f(min, min, min);
	rval *= 256 / delta;
	if (rval.x > 255)dataByte[tId].r = 255;
	else if (rval.x < 0)dataByte[tId].r = 0;
	else dataByte[tId].r = (unsigned char)rval.x;
	if (rval.y > 255)dataByte[tId].g = 255;
	else if (rval.y < 0)dataByte[tId].g = 0;
	else dataByte[tId].g = (unsigned char)rval.y;
	if (rval.z > 255)dataByte[tId].b = 255;
	else if (rval.z < 0)dataByte[tId].b = 0;
	else dataByte[tId].b = (unsigned char)rval.z;
}

void displayCudaError(size_t id = 0) {
#ifdef __GPUDEBUG
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	std::cout << "#" << id <<"  "<< hipGetErrorName(err)<< std::endl;
	if (err != hipError_t::hipSuccess) {
		int x;
		std::cin >> x;
	}
#else

#endif
}

void generateGPUDisplatData(colorBYTE** data , camera cam) {
	displayCudaError(11);
	hipMalloc(data, sizeof(colorBYTE) * cam.sc.resX * cam.sc.resY);
	displayCudaError(10);
}

void renderIntermediate(camera cam,colorBYTE* DataByte, meshShaded* meshS, meshConstrained* meshC, size_t noTrs) {
	displayCudaError(9);
	linearMath::linef* rays;
	displayCudaError(8);
	hipMalloc(&rays, sizeof(linearMath::linef) * cam.sc.resX * cam.sc.resY);
	displayCudaError(7);
	initRays << <blockNo(cam.sc.resX * cam.sc.resY), threadNo >> > (cam.sc.resX, cam.sc.resY, cam.vertex, cam.sc.screenCenter - cam.sc.halfRight + cam.sc.halfUp, cam.sc.halfRight * 2, cam.sc.halfUp * -2, rays);
	displayCudaError(1);
	skyboxCPU defaultShader(color(0, 0, 128), color(-200, -200, -200), color(150, 0, 0), color(0, 0, 64), color(0, 0, 64), color(0, 0, 64));
	//solidColCPU defaultShader(color(0, 0, 0));
	displayCudaError(2);
	color* Data;
	hipMalloc(&Data, sizeof(color) * cam.sc.resX * cam.sc.resY);
	displayCudaError(3);
	getIntersections << <blockNo(cam.sc.resX * cam.sc.resY), threadNo >> > (rays, cam.sc.resX * cam.sc.resY, meshS, meshC, noTrs, Data, defaultShader.getGPUPtr());
	displayCudaError(4);
	getByteColor << <blockNo(cam.sc.resX * cam.sc.resY), threadNo >> > (Data, DataByte, 0, 256, cam.sc.resX * cam.sc.resY);
	displayCudaError(5);
	hipFree(Data);
	hipFree(rays);
	displayCudaError(6);
}

void cpyData(colorBYTE* data , BYTE * displayData, camera cam) {
	hipDeviceSynchronize();
	displayCudaError(12);
	hipMemcpy(displayData, data, sizeof(colorBYTE) * cam.sc.resX * cam.sc.resY, hipMemcpyKind::hipMemcpyDeviceToHost);
	displayCudaError(13);
	hipFree(data);
	displayCudaError(14);
}

void Render(camera cam,BYTE *data, meshShaded * meshS , meshConstrained * meshC , size_t noTrs) {
	colorBYTE* displayData;
	generateGPUDisplatData(&displayData, cam);
	renderIntermediate(cam, displayData, meshS, meshC, noTrs);
	cpyData(displayData, data, cam);
}


void graphicalWorld::render(camera cam, BYTE* data) {


	bool updated=false;
	meshShaded* devPtr = meshS->getDevice(&updated);
	if(updated){
		initMesh<<<blockNo(meshS->getNoElements()),threadNo>>>(devPtr, meshC->getDevice(), meshS->getNoElements());
	}
	Render(cam, data, meshS->getDevice(), meshC->getDevice(), meshS->getNoElements());

}

void graphicalWorld::render(camera cam, BYTE* data, std::function<void()> drawCall) {
	std::thread draw(drawCall);
	renderPartial(cam);
	draw.join();
	copyData(cam, data);
}

void graphicalWorld::renderPartial(camera cam) {
	
	bool updated = false;
	meshShaded* devPtr = meshS->getDevice(&updated);
	if (updated) {
		initMesh << <blockNo(meshS->getNoElements()), threadNo >> > (devPtr, meshC->getDevice(), meshS->getNoElements());
	}
	if (tempData != nullptr) {
		hipDeviceSynchronize();
		hipFree(tempData);
		tempData = nullptr;
		displayCudaError(16);
	}
	generateGPUDisplatData(&tempData, cam);
	renderIntermediate(cam, tempData, meshS->getDevice(), meshC->getDevice(), meshS->getNoElements());
	//Render(cam, data, meshS->getDevice(), meshC->getDevice(), meshS->getNoElements());
}

void graphicalWorld::copyData(camera cam, BYTE* data) {
	if (tempData != nullptr) {
		cpyData(tempData, data, cam);
		tempData = nullptr;
	}
}



///ADV GRAPHICS WORLD CUDA_functions

namespace ADVRTX {

	__global__
	void initRays(short xResReq,short yResReq,short xRes, short yRes, vec3f vertex, vec3f topLeft, vec3f right, vec3f down, linearMath::linef* rays) {
		size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
		if (tId >= (xResReq * yResReq))return;

		short x, short y;
		x = tId % xResReq;
		y = tId / xResReq;

		rays[tId].setRaw_s(vertex, vec3f::subtract(vec3f::add(topLeft, vec3f::add(vec3f::multiply(right, (x + 0.5) / xRes), vec3f::multiply(down, (y + 0.5) / yRes))), vertex));
	}


	//get intersections only for rays on grid points
	//rays ptr to all rays
	//noGPts no of grid points include extra ones
	//xbatch , x separation between grid points , includeing extra x pt
	//ybatch , y separation between two grid pts(including the 1 extra pt) * image width
	__global__
	void getIntersections(linearMath::linef* rays, size_t noGPts,unsigned short noXGridPts, unsigned short xBatch , unsigned short yBatch, meshShaded* trs, meshConstrained* collTrs, size_t noTrs, color* displayData, chromaticShader* defaultShader) {
		size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
		if (tId >= noGPts)return;

		unsigned short xCoord = tId % noXGridPts;
		unsigned short yCoord = tId / noXGridPts;

		tId = xCoord * xBatch + yCoord * yBatch;

		getIntersectionsInternal(rays + tId, trs, collTrs, noTrs, displayData + tId, defaultShader);

	}


	__global__
		void getByteColor(color* data, colorBYTE* dataByte, float min, float delta,unsigned short xRes,unsigned short yRes , unsigned short xReqRes ) {
		size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
		if (tId >= xRes * yRes)return;
		
		size_t baseId = (tId / xRes) * xReqRes + (tId % xRes);
		color rval = data[baseId];
		rval -= vec3f(min, min, min);
		rval *= 256 / delta;

		if (rval.x > 255)dataByte[tId].r = 255;
		else if (rval.x < 0)dataByte[tId].r = 0;
		else dataByte[tId].r = (unsigned char)rval.x;
		if (rval.y > 255)dataByte[tId].g = 255;
		else if (rval.y < 0)dataByte[tId].g = 0;
		else dataByte[tId].g = (unsigned char)rval.y;
		if (rval.z > 255)dataByte[tId].b = 255;
		else if (rval.z < 0)dataByte[tId].b = 0;
		else dataByte[tId].b = (unsigned char)rval.z;
	}


	__global__
	void doubleResX() {

	}


}

///ADV GRAPHICS WORLD functions 
graphicalWorldADV::graphicalWorldADV(commonMemory<meshShaded>* meshPtr, unsigned short xResolution, unsigned short yResolution, unsigned char xIters, unsigned char yIters) {
	meshS = meshPtr;
	meshC = new commonMemory<meshConstrained>(meshS->getNoElements(), commonMemType::deviceOnly);
	xDoublingIterations = xIters;
	yDoublingIterations = yIters;
	xRes = xResolution;
	yRes = yResolution;

	//calculate multiplication factor
	mulFacX = 1;
	for (unsigned short i = 0; i < xIters; ++i)mulFacX *= 2;
	mulFacY = 1;
	for (unsigned short i = 0; i < yIters; ++i)mulFacY *= 2;

	unsigned short rSamplesX = (xRes / mulFacX);
	if (xRes % mulFacX != 0) rSamplesX++;
	unsigned short rSamplesY = (yRes / mulFacY);
	if (yRes % mulFacY != 0) rSamplesY++;


	xResReq = rSamplesX * mulFacX + 1;
	yResReq = rSamplesY * mulFacY + 1;

	gridX = rSamplesX + 1;
	gridY = rSamplesY + 1;

#ifdef __GPUDEBUG
	std::cout << "actual res   = " << xRes << " , " << yRes << std::endl;
	std::cout << "required res = " << xResReq << " , " << yResReq << std::endl;
	std::cout << "grid res     = " << gridX << " , " << gridY << std::endl;
	std::cout << "m Factors    = " << mulFacX << " , " << mulFacY << std::endl;
	std::cout << "d iterations = " << xDoublingIterations << " , " << yDoublingIterations << std::endl;
#endif

	hipMalloc(&redundancyData, sizeof(redundancyData) * xResReq * yResReq);
	hipMalloc(&rays, sizeof(linearMath::linef) * xResReq * yResReq);
	hipMalloc(&tempData, sizeof(color) * xResReq * yResReq);
	hipMalloc(&actualResData, sizeof(colorBYTE) * xRes * yRes);
}

graphicalWorldADV::~graphicalWorldADV() {

	//delete data created
	delete meshC;
	hipFree(redundancyData);
	hipFree(rays);
	hipFree(tempData);
	hipFree(actualResData);
}


void graphicalWorldADV::render(camera cam, BYTE* data) {
	displayCudaError(0);
	//init mesh
	bool updated = false;
	meshShaded* devPtr = meshS->getDevice(&updated);
	if (updated) {
		initMesh << <blockNo(meshS->getNoElements()), threadNo >> > (devPtr, meshC->getDevice(), meshS->getNoElements());
	}
	displayCudaError(1);

	//init rays
	ADVRTX::initRays<<<blockNo(xResReq * yResReq), threadNo >>>(xResReq, yResReq, xRes, yRes, cam.vertex, cam.sc.screenCenter - cam.sc.halfRight + cam.sc.halfUp, cam.sc.halfRight * 2, cam.sc.halfUp * -2, rays);
	//skyboxCPU defaultShader(color(0, 0, 128), color(-200, -200, -200), color(150, 0, 0), color(0, 0, 64), color(0, 0, 64), color(0, 0, 64));
	solidColCPU defaultShader(color(255, 255, 255));
	displayCudaError(2);
	//do rtx
	ADVRTX::getIntersections << <blockNo(gridX * gridY), threadNo >> > (rays, gridX * gridY, gridX, mulFacX, mulFacY * xResReq, meshS->getDevice(), meshC->getDevice(), meshC->getNoElements(), tempData, defaultShader.getGPUPtr());
	displayCudaError(3);
	ADVRTX::getByteColor<<<blockNo(xRes*yRes), threadNo >>>(tempData, actualResData, 0, 255, xRes, yRes, xResReq);
	displayCudaError(4);
	//copy data
	hipDeviceSynchronize();
	hipMemcpy(data, actualResData, sizeof(colorBYTE) * xRes * yRes, hipMemcpyKind::hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	displayCudaError(5);
}