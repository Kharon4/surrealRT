#include "hip/hip_runtime.h"
#include "rendering.cuh"
#include <iostream>

#define threadNo 1024
#define blockNo(Threads) Threads/threadNo

struct intersectionParam{
	short camX, camY;
	double lambda;
	vec3d pt;
	mesh* mesh;
	meshConstrained* meshConstrained;
};


__global__
void initRays(short xRes , short yRes , vec3d vertex , vec3d topLeft , vec3d right , vec3d down , linearMathD::line * rays) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= (xRes * yRes))return;
	
	short x, short y;
	x = tId % xRes;
	y = tId / yRes;

	rays[tId].setRaw_s(vertex, vec3d::subtract(vec3d::add(topLeft, vec3d::add(vec3d::multiply(right, (x + 0.5) / xRes), vec3d::multiply(down, (y + 0.5) / yRes))), vertex));
}

__device__ __host__ void calculateMeshConstraints(mesh* Mesh , meshConstrained *meshC){
	vec3d plNormal = vec3d::cross(Mesh->pts[1] - Mesh->pts[0], Mesh->pts[2] - Mesh->pts[0]);
	meshC->planeNormal = plNormal;
	meshC->sidePlaneNormals[0] = vec3d::cross(plNormal, Mesh->pts[1] - Mesh->pts[0]);
	meshC->sidePlaneNormals[1] = vec3d::cross(plNormal, Mesh->pts[2] - Mesh->pts[1]);
	meshC->sidePlaneNormals[2] = vec3d::cross(plNormal, Mesh->pts[0] - Mesh->pts[2]);
}

__global__
void initMesh(mesh* Mesh, meshConstrained* meshC, size_t noOfThreads) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= noOfThreads)return;
	calculateMeshConstraints(Mesh + tId, meshC + tId);
}

__global__
void getIntersections(linearMathD::line * rays , mesh ** intersections , size_t noRays) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= noRays)return;
	intersections[tId] = nullptr;
}

__global__
void shadeKernel(mesh** interactions,linearMathD::line* rays, color* data, chromaticShader** defaultShader , size_t maxNo) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= maxNo)return;
	shaderData df;
	df.dr = rays[tId].getDr();
	if (interactions[tId] == nullptr)data[tId] = (*defaultShader)->shade(df);
}

__global__
void getByteColor(color* data, colorBYTE* dataByte, float min, float delta, size_t noThreads) {
	size_t tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId >= noThreads)return;
	color rval = data[tId];
	rval -= vec3f(min, min, min);
	rval *= 256 / delta;
	if (rval.x > 255)dataByte[tId].r = 255;
	else if (rval.x < 0)dataByte[tId].r = 0;
	else dataByte[tId].r = (unsigned char)rval.x;
	if (rval.y > 255)dataByte[tId].g = 255;
	else if (rval.y < 0)dataByte[tId].g = 0;
	else dataByte[tId].g = (unsigned char)rval.y;
	if (rval.z > 255)dataByte[tId].b = 255;
	else if (rval.z < 0)dataByte[tId].b = 0;
	else dataByte[tId].b = (unsigned char)rval.z;
}

__global__
void createShader(chromaticShader ** ptr){
	color c,down,red;
	c.x = -255;
	c.y = 100;
	c.z = 200;
	down.x = -255;
	down.y = 0;
	down.z = 0;
	red.x = 700;
	*ptr = new skybox(c,down,red,down,down,down);
}

__global__
void deleteShader(chromaticShader** ptr)
{
	delete (*ptr);
}

void displayCudaError() {
	hipDeviceSynchronize();
	std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
}

void render(camera cam,BYTE *data) {
	linearMathD::line * rays;
	hipMalloc(&rays, sizeof(linearMathD::line) * cam.sc.resX * cam.sc.resY);
	initRays<<<threadNo , blockNo(cam.sc.resX*cam.sc.resY)>>>(cam.sc.resX, cam.sc.resY, cam.vertex, cam.sc.screenCenter - cam.sc.halfRight + cam.sc.halfUp, cam.sc.halfRight * 2, cam.sc.halfUp * -2, rays);
	mesh** intersections;
	hipMalloc(&intersections, sizeof(mesh*) * cam.sc.resX * cam.sc.resY);
	getIntersections << <threadNo, blockNo(cam.sc.resX * cam.sc.resY) >> > (rays, intersections, cam.sc.resX * cam.sc.resY);
	chromaticShader** sc;
	hipMalloc(&sc, sizeof(chromaticShader*));
	createShader<<<1,1>>>(sc);
	color* Data;
	hipMalloc(&Data, sizeof(color) * cam.sc.resX * cam.sc.resY);
	shadeKernel << <threadNo, blockNo(cam.sc.resX * cam.sc.resY) >> > (intersections,rays, Data, sc, cam.sc.resX * cam.sc.resY);
	colorBYTE *DataByte;
	hipMalloc(&DataByte, sizeof(colorBYTE) * cam.sc.resX * cam.sc.resY);
	getByteColor << <threadNo, blockNo(cam.sc.resX * cam.sc.resY) >> > (Data, DataByte, 0, 256, cam.sc.resX * cam.sc.resY);
	hipMemcpy(data, DataByte, sizeof(colorBYTE) * cam.sc.resX * cam.sc.resY, hipMemcpyKind::hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(DataByte);
	hipFree(Data);
	deleteShader << <1, 1 >> > (sc);
	hipFree(sc);
	hipFree(intersections);
	hipFree(rays);
}