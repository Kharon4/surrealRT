#define _INSIDE_commonMemory_Header 1
#include "cudaRelated/commonMemory.cuh"

#include "hip/hip_runtime.h"
#include ""


template <typename T>
commonMemory<T>::commonMemory(size_t Size, commonMemType Type) {
	noElements = Size;
	size = sizeof(T) * Size;
	type = Type;
	if (size == 0)return;

	if (type != commonMemType::deviceOnly)
		hostPtr = new unsigned char[size];

	if (type != commonMemType::hostOnly)
		hipMalloc(&devicePtr, size);
}

template <typename T>
size_t commonMemory<T>::getNoElements() {
	return noElements;
}

template <typename T>
T* commonMemory<T>::getHost(bool* OUTupdated) {
	if (OUTupdated != nullptr)*OUTupdated = false;

	if (type == deviceOnly)return (T*)nullptr;
	if (!hostUpdated && type == both) {
		hipMemcpy(hostPtr, devicePtr, size, hipMemcpyKind::hipMemcpyDeviceToHost);
		hostUpdated = true;
		if (OUTupdated != nullptr)*OUTupdated = true;
	}
	return (T*)hostPtr;
}

template <typename T>
T* commonMemory<T>::getDevice(bool* OUTupdated) {
	if (OUTupdated != nullptr)*OUTupdated = false;

	if (type == hostOnly)return (T*)nullptr;
	if (hostUpdated && type == both) {
		hipMemcpy(devicePtr, hostPtr, size, hipMemcpyKind::hipMemcpyHostToDevice);
		hostUpdated = false;
		if (OUTupdated != nullptr)*OUTupdated = true;
	}
	return (T*)devicePtr;
}


template <typename T>
void commonMemory<T>::changeMemType(commonMemType newType) {
	//check if new type and old type r same.
	if (newType == type)return;
	
	//if !both create space
	if (type != commonMemType::both) {
		if (type == commonMemType::hostOnly) {
			//create device mem
			hipMalloc(&devicePtr, size);
			//copy data
			hipMemcpy(devicePtr, hostPtr, size, hipMemcpyKind::hipMemcpyHostToDevice);
		}
		else {
			//create host mem
			hostPtr = new unsigned char[size];
			//copy data
			hipMemcpy(hostPtr, devicePtr, size, hipMemcpyKind::hipMemcpyDeviceToHost);
		}
	}
	else {
		//update the correct side;
		getHost();
		getDevice();
	}

	//delete
	if (newType == commonMemType::hostOnly) {
		hostUpdated = true;
		//delete device
		if (devicePtr != nullptr)hipFree(devicePtr);
		devicePtr = nullptr;
	}
	else if (newType == commonMemType::deviceOnly) {
		hostUpdated = false;
		//delete host
		if (hostPtr != nullptr)delete[] hostPtr;
		hostPtr = nullptr;
	}

	//perform conversion
	type = newType;
}

template <typename T>
commonMemType commonMemory<T>::getMemType() {
	return type;
}


template <typename T>
void commonMemory<T>::operator= (const commonMemory<T>& other) {
	//delete stuff
	if (hostPtr != nullptr)delete[] hostPtr;
	if (devicePtr != nullptr)hipFree(devicePtr);

	//copy stuff
	hostPtr = other.hostPtr;
	devicePtr = other.devicePtr;
	hostUpdated = other.hostUpdated;
	noElements = other.noElements;
	size = other.size; 
	type = other.type;

	//dissable other
	other.hostPtr = nullptr;
	other.devicePtr = nullptr;

}

template <typename T>
commonMemory<T>::~commonMemory() {
	if (hostPtr != nullptr)delete[] hostPtr;
	if (devicePtr != nullptr)hipFree(devicePtr);

}