#define _INSIDE_commonMemory_Header 1
#include "cudaRelated/commonMemory.cuh"

#include "hip/hip_runtime.h"
#include ""

template <typename T>
commonMemory<T>::commonMemory(size_t Size, commonMemType Type) {
	size = sizeof(T) * Size;
	type = Type;
	if (size == 0)return;

	if (type != commonMemType::deviceOnly)
		hostPtr = new unsigned char[size];

	if (type != commonMemType::hostOnly)
		hipMalloc(&devicePtr, size);
}

template <typename T>
T* commonMemory<T>::getHost() {
	if (type == deviceOnly)return (T*)nullptr;
	if (!hostUpdated && type == both) {
		hipMemcpy(hostPtr, devicePtr, size, hipMemcpyKind::hipMemcpyDeviceToHost);
		hostUpdated = true;
	}
	return (T*)hostPtr;
}

template <typename T>
T* commonMemory<T>::getDevice() {
	if (type == hostOnly)return (T*)nullptr;
	if (hostUpdated && type == both) {
		hipMemcpy(devicePtr, hostPtr, size, hipMemcpyKind::hipMemcpyHostToDevice);
		hostUpdated = false;
	}
	return (T*)devicePtr;
}

template <typename T>
commonMemory<T>::~commonMemory() {
	if (hostPtr != nullptr)delete[] hostPtr;
	if (devicePtr != nullptr)hipFree(devicePtr);

}