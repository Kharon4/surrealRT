#define _INSIDE_commonMemory_Header 1
#include "cudaRelated/commonMemory.cuh"

#include "hip/hip_runtime.h"
#include ""

template <typename T>
commonMemory<T>::commonMemory(size_t Size, commonMemType Type) {
	noElements = Size;
	size = sizeof(T) * Size;
	type = Type;
	if (size == 0)return;

	if (type != commonMemType::deviceOnly)
		hostPtr = new unsigned char[size];

	if (type != commonMemType::hostOnly)
		hipMalloc(&devicePtr, size);
}

template <typename T>
size_t commonMemory<T>::getNoElements() {
	return noElements;
}

template <typename T>
T* commonMemory<T>::getHost(bool* OUTupdated) {
	if (OUTupdated != nullptr)*OUTupdated = false;

	if (type == deviceOnly)return (T*)nullptr;
	if (!hostUpdated && type == both) {
		hipMemcpy(hostPtr, devicePtr, size, hipMemcpyKind::hipMemcpyDeviceToHost);
		hostUpdated = true;
		if (OUTupdated != nullptr)*OUTupdated = true;
	}
	return (T*)hostPtr;
}

template <typename T>
T* commonMemory<T>::getDevice(bool* OUTupdated) {
	if (OUTupdated != nullptr)*OUTupdated = false;

	if (type == hostOnly)return (T*)nullptr;
	if (hostUpdated && type == both) {
		hipMemcpy(devicePtr, hostPtr, size, hipMemcpyKind::hipMemcpyHostToDevice);
		hostUpdated = false;
		if (OUTupdated != nullptr)*OUTupdated = true;
	}
	return (T*)devicePtr;
}

template <typename T>
commonMemory<T>::~commonMemory() {
	if (hostPtr != nullptr)delete[] hostPtr;
	if (devicePtr != nullptr)hipFree(devicePtr);

}